#include "hip/hip_runtime.h"
#include <thrust/tuple.h>

#include <ATen/Dispatch.h>
#include "../common/Loops.h"
#include <ATen/native/cuda/Loops.cuh>

#include "../common/complex.h" // NOLINT(modernize-deprecated-headers)


#define DEF_ROOTS(N) template <typename scalar_t, typename T=complex<scalar_t>> __host__ __device__ std::array<T, N> roots##N##_kernel


DEF_ROOTS(2)(T b, T c) {
    if (not (isfinite(b) and isfinite(c)))
        return {std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN()};
    return {(-b - sqrt(b*b - ltrl(4)*c)) / ltrl(2),
            (-b + sqrt(b*b - ltrl(4)*c)) / ltrl(2)};
}


DEF_ROOTS(3)(T b, T c, T d) {
    if (not (isfinite(b) and isfinite(c) and isfinite(d)))
        return {std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN()};
    auto D0 = b*b - 3*c,
         D1 = ltrl(2)*b*b*b - ltrl(9)*b*c + ltrl(27)*d;
    if (D0 == ltrl(0) and D1 == ltrl(0))
        return {-b/ltrl(3), -b/ltrl(3), -b/ltrl(3)};

    auto D2 = sqrt(D1*D1 - ltrl(4)*D0*D0*D0),
         C = D1 + D2;
    if (C == ltrl(0)) C = D1 - D2;
    C = pow(C / ltrl(2), ltrl((1./3.)));
    auto cr1 = T(-0.5, -sqrt(3)/2);
    return {-(b + C + D0/C) / ltrl(3),
            -(b + C*cr1 + D0/(C*cr1)) / ltrl(3),
            -(b + C/cr1 + D0/(C/cr1)) / ltrl(3)};
}


DEF_ROOTS(4)(T b, T c, T d, T e) {
    if (not (isfinite(b) and isfinite(c) and isfinite(d) and isfinite(e)))
        return {std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN(),
                std::numeric_limits<T>::quiet_NaN()};
    auto twop = (ltrl(8)*c - ltrl(3)*b*b) / ltrl(4),
         q = (b*b*b - ltrl(4)*b*c + ltrl(8)*d) / ltrl(8),
         D0 = c*c - ltrl(3)*b*d + ltrl(12)*e,
         D1 = ltrl(2)*c*c*c - ltrl(9)*b*c*d + ltrl(27)*b*b*e + ltrl(27)*d*d - ltrl(72)*c*e;
    T Q;
    if (D0 == ltrl(0)) {
        if (D1 == ltrl(0)) {
            if (q == ltrl(0)) return {ltrl(0), ltrl(0), ltrl(0), ltrl(0)};
            auto x0 = (-ltrl(72)*e + ltrl(10)*c*c - ltrl(3)*b*b*c) / ltrl(9) / q,
                 x1 = -b - ltrl(3)*x0;
            return {x0, x0, x0, x1};
        } else Q = pow(D1, ltrl(1./3.));
    } else Q = pow(D1 + sqrt(D1*D1 - ltrl(4) * D0*D0*D0), ltrl(1./3.)) / cbrt(ltrl(2));
    auto _s2 = ((Q + D0 / Q) - twop) / ltrl(12);
    T s2;
    if (_s2 == ltrl(0)) {
        Q *= T(-0.5, -sqrt(3)/2);
        s2 = ((Q + D0 / Q) - twop) / ltrl(12);
    } else s2 = _s2;
    auto S = sqrt(s2),
         pmp = sqrt(ltrl(-4)*s2 - twop + q/S) / ltrl(2),
         pmm = sqrt(ltrl(-4)*s2 - twop - q/S) / ltrl(2),
         mb4apS = -b / ltrl(4) + S,
         mb4amS = -b / ltrl(4) - S;
    return {mb4amS - pmp, mb4amS + pmp, mb4apS - pmm, mb4apS + pmm};
}


#define ROOTS_IMPL(N, VARSPEC, VARNAMES, ...) \
    if (iter.device(0).is_cpu())              \
        cpu_kernel_multiple_outputs(iter, roots##N##_kernel<scalar_t>); \
    else at::native::gpu_kernel_multiple_outputs(                     \
        iter, []GPU_LAMBDA VARSPEC -> thrust::tuple<__VA_ARGS__> {      \
            return c10::guts::apply(thrust::make_tuple<__VA_ARGS__>, roots##N##_kernel<scalar_t>VARNAMES);});


template <int n> void roots_impl(at::TensorIteratorBase& iter) {
    TORCH_CHECK(iter.device(0).is_cpu() or iter.device(0).is_cuda(),
                "\"roots4_kernel_cuda\" only implemented on CPU and cuda.")

    AT_DISPATCH_FLOATING_TYPES(toValueType(iter.common_dtype()), "roots", [&] {
        using T = complex<scalar_t>;

        // TODO: Why doesn't if constexpr work with more than one ROOTS_IMPl?!...
        switch (n) {
            case 2: ROOTS_IMPL(2, (T b, T c), (b, c), T, T); break;
            case 3: ROOTS_IMPL(3, (T b, T c, T d), (b, c, d), T, T, T); break;
            case 4: ROOTS_IMPL(4, (T b, T c, T d, T e), (b, c, d, e), T, T, T, T); break;
            default: TORCH_CHECK(false, "\"roots\" only implemented for orders 2, 3, 4.")
        }
    });
}

template void roots_impl<2>(at::TensorIteratorBase& iter);
template void roots_impl<3>(at::TensorIteratorBase& iter);
template void roots_impl<4>(at::TensorIteratorBase& iter);
