#include "hip/hip_runtime.h"
#include "special.cuh"

DEFINE_COMPLEX_FUNCTION(loggamma, (z)) {
    if (is_int(z) and is_real_nonpositive(z)) return numeric_limits<T>::infinity();

    T res;
    bool doconj = z.imag() < 0;
    if (doconj) z = conj(z);

    if (z.real() < -14 or z.imag() < -14) {
        res = ltrl(LOGPI) - loggamma<scalar_t>(1-z) - (
                (z.imag() > 36.7 or z.imag() < -36.7) ?
                ltrl(M_PI) * z.imag() - ltrl(0.6931471805599453094) + T(0, M_PI * (0.5 - z.real())) :
                log(sin(z-floor(z.real()))) - T(0, M_PI * floor(z.real()))
        );
    } else {
        T c = 1;
        scalar_t a = 0;

        if (z.real() < 14)
            while (z.real() < 14) {
                c *= z;
                a += arg(z);
                z += 1;
            }

        res = (z - ltrl(0.5)) * log(z) - z + ltrl(LOGSQRT2PI) - T(log(abs(c)), a);

        if (abs(z) < 1e8) {
            auto w = ltrl(1) / (z*z);
            // TODO: macro-ify
            res += (((((ltrl(-1.9175269175269175269175269175269175269175e-3) * w
                      + ltrl(8.4175084175084175084175084175084175084175E-4)) * w
                      + ltrl(-5.9523809523809523809523809523809523809524e-4)) * w
                      + ltrl(7.9365079365079365079365079365079365079365e-4)) * w
                      + ltrl(-2.7777777777777777777777777777777777777778e-3)) * w
                      + ltrl(8.3333333333333333333333333333333333333333e-2)) / z;
        }
    }

    return doconj ? conj(res) : res;
}
