#include "hip/hip_runtime.h"
#include "elliptic.cuh"


DEFINE_COMPLEX_FUNCTION(elliprc, (x, y)) {
    if (isinf(x) or isinf(y)) return ltrl(1) / (x*y);
    if (not y) return numeric_limits<T>::infinity();
    if (not x) return ltrl(M_PI_2) / sqrt(y);

    // principal value
    if (not y.imag() and y.real() < 0) return sqrt(x / (x-y)) * elliprc<scalar_t>(x-y, -y);

    // TODO: handle x=y in elliprc better
    if (abs(sqrt(1-x/y)) < max((scalar_t) 1e-3, 100*sqrt(numeric_limits<scalar_t>::epsilon())))
        return (ltrl(7./6.) - (x/y) / ltrl(6)) / sqrt(y);

    // Formally, have to T(1), but this error most times "cancels" with wrong
    // continuity of acos...
    // return acos(sqrt(x) / sqrt(y)) / (sqrt(1-x/y) * sqrt(y));

    auto v = sqrt(x) / sqrt(y), _1vv = T(1) - v*v, s1vv = sqrt(_1vv), lv = v*TIMAG + s1vv;
    auto ac = ltrl(M_PI_2) + TIMAG*log(lv);
    auto _1xy = T(1)-x/y, s1xy = sqrt(_1xy);
    //
    // // printf("v=%.20e+i%.20e, v^2=%.20e+i%.20e, 1-v^2=%.20e+i%.20e\n",
    // //        v.real(), v.imag(),
    // //        (v*v).real(), (v*v).imag(),
    // //        _1vv.real(), _1vv.imag());
    // // printf("sqrt(%.4e+i%.4e)=%.4e+i%.4e\n", (T(1)-v*v).real(), (1-v*v).imag(), s1vv.real(), s1vv.imag());
    // // printf("log(%.4e+i%.4e) = %.4e+i%.4e\n", lv.real(), lv.imag(), log(lv).real(), log(lv).imag());
    // // printf("acos(%.4e+i%.4e) = %.4e+i%.4e\n", v.real(), v.imag(), ac.real(), ac.imag());
    // // printf("sqrt(%.4e+i%.4e) = %.4e+i%.4e\n", _1xy.real(), _1xy.imag(), s1xy.real(), s1xy.imag());
    //
    return ac / (s1xy * sqrt(y));
}
